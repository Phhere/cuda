#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <limits.h>
#include "cuda_kernel.cuh"

int solveProblem(const int argc, const char* argv[]){
	hipError_t return_value;
	if(argc == 2){
		hipEvent_t start, stop;
		float time;
		int threads = atoi(argv[1]);
		hipEventCreate(&start);
		hipEventCreate(&stop);
		// Everything runs on stream 0
		hipEventRecord(start);
		kernel<<<1,threads>>>();
		hipDeviceSynchronize();
		hipEventRecord(stop);
		hipEventSynchronize(stop);
	    return_value = hipGetLastError();
	    if(return_value != hipSuccess){
	    	printf("Error in Kernel\n");
	    	printf("%s\n",hipGetErrorString(return_value));
	    	return -1;
	    }
	    hipEventElapsedTime(&time, start, stop);
	    printf ("Time for the kernel: %f ms\n", time);
		return 0;
	}
	else{
		printf("parameter required\n");
		return -1;
	}
}

int main(const int argc, const char* argv[]){
	int devices;
	hipError_t return_value;
	return_value = hipGetDeviceCount(&devices);
	if(return_value != hipSuccess){
		printf("Could not get device count\n");	
		return -1;
	}

	if(devices > 0){
		printf("%d devices found\n",devices);
		for(int device = 0; device < devices; device++){
			hipDeviceProp_t device_info;
			hipGetDeviceProperties(&device_info, device);
			printf("Name: %s\n",device_info.name);
			printf("max. Memory: %.0f MB\n",(double)device_info.totalGlobalMem/(double)(1024*1024));
			printf("max. Threads per Block: %d\n", device_info.maxThreadsPerBlock);
			printf("max. Blocks per Grid: %d,%d,%d\n", device_info.maxGridSize[0], device_info.maxGridSize[1],device_info.maxGridSize[2]);
			printf("\n");
		}
		return solveProblem(argc, argv);
	}
	else{
		printf("No CUDA cards found\n");
		return -1;
	}

}