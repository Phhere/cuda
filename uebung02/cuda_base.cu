#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <limits.h>
#include "cuda_kernel.cuh"

int solveProblem(const int argc, const char* argv[]){
	hipError_t return_value;
	if(argc == 2){
		hipEvent_t start, stop;
		float time;
		int vectorlength = atoi(argv[1]);
		hipEventCreate(&start);
		hipEventCreate(&stop);
		// Everything runs on stream 0
		hipEventRecord(start);

		/* Start Program */

        size_t size = sizeof(int) * vectorlength;
        int *hosta = (int*) malloc(size);
        int *hostb = (int*) malloc(size);
        int *hostc = (int*) malloc(size);
        // maybe null?

        int *deva, *devb, *devc;

        hipMalloc((void**) &deva, size);
        hipMalloc((void**) &devb, size);
        hipMalloc((void**) &devc, size);
        // maybe null?

        for(int i = 0; i < vectorlength; i++) {
            hosta[i] = i;
            hostb[i] = i*i;
        }

        // Copy vectors to device
        hipMemcpy(deva, hosta, size, hipMemcpyHostToDevice);
        hipMemcpy(devb, hostb, size, hipMemcpyHostToDevice);

        // Calculate blocksize and threadnumber
        int blocksPerGrid = 1;
        int threadsPerBlock = vectorlength;

        if (vectorlength > 1024) {
            blocksPerGrid = (int) ceil(vectorlength / 1024.0);
            threadsPerBlock = 1024;
        }

        // Kernel time!
		kernel<<<blocksPerGrid, threadsPerBlock>>>(deva, devb, devc);

        // Copy results back to host
        hipMemcpy(hostc, devc, size, hipMemcpyDeviceToHost);

        // Print results
        for (int i = 0; i < vectorlength; ++i) {
            printf("C[%d] = %d\n", i, hostc[i]);
        }

        hipFree(deva);
        hipFree(devb);
        hipFree(devc);

        free(hosta);
        free(hostb);
        free(hostc);

        /* End Program */

		hipDeviceSynchronize();
		hipEventRecord(stop);
		hipEventSynchronize(stop);
	    return_value = hipGetLastError();
	    if(return_value != hipSuccess){
	    	printf("Error in Kernel\n");
	    	printf("%s\n",hipGetErrorString(return_value));
	    	return -1;
	    }
	    hipEventElapsedTime(&time, start, stop);
	    printf ("Time for the kernel: %f ms\n", time);
		return 0;
	} else {
		printf("parameter required\n");
		return -1;
	}
}

int main(const int argc, const char* argv[]){
	int devices;
	hipError_t return_value;
	return_value = hipGetDeviceCount(&devices);
	if(return_value != hipSuccess){
		printf("Could not get device count\n");
		return -1;
	}

	if(devices > 0){
		printf("%d devices found\n",devices);
		for(int device = 0; device < devices; device++){
			hipDeviceProp_t device_info;
			hipGetDeviceProperties(&device_info, device);
			printf("Name: %s\n",device_info.name);
			printf("max. Memory: %.0f MB\n",(double)device_info.totalGlobalMem/(double)(1024*1024));
			printf("max. Threads per Block: %d\n", device_info.maxThreadsPerBlock);
			printf("max. Blocks per Grid: %d,%d,%d\n", device_info.maxGridSize[0], device_info.maxGridSize[1],device_info.maxGridSize[2]);
			printf("\n");
		}
		return solveProblem(argc, argv);
	}
	else{
		printf("No CUDA cards found\n");
		return -1;
	}

}
